#include "hip/hip_runtime.h"
// JNI
#include <jni.h>
#include "gpu_GPUmm.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <stdio.h>
#include <cassert>
#include <iostream>
#include <fstream>
#include <cstring>
#include <sstream>
#include <sys/time.h>
#include <string>
#include <cstdlib>
#include <cmath>

using namespace std;

// ====== Optimiations ====
#define REGULATE_GPU  // ON
//#define OPT_TRIANGULAR_MM      // OFF
#define OPT_SPARSE_MATRIX        // OFF
#define OPT_EARLY_TERMINATION  // ON


// ====== Vars ======
#define THREADS_PER_BLOCK 512
#define REGULATE_BATCH 1000

#define MAX_N 30000ul
//#define MAX_N 16384ul
#define MAX_NNZ ((MAX_N) * 20)

// sparse matrix optimization
#ifdef OPT_SPARSE_MATRIX
  #define MAGIC_SPARSE_THRESHOLD1 0.01
  #define MAGIC_SPARSE_THRESHOLD2 12
#else
  #define MAGIC_SPARSE_THRESHOLD1 0
  #define MAGIC_SPARSE_THRESHOLD2 0
#endif

// early termination optimization
#define MAGIC_EARLY_TERMINATION_THRESHOLD 256





const float alpha = 1.0;
const float beta = 0.0;

// TODO: check which API needs sync
float *gpu_m, *gpu_m2, *gpu_csr_val;
int *gpu_nnz_row, *gpu_csr_rowptr, *gpu_csr_colind;

hipblasHandle_t handle_c;
hipsparseHandle_t handle_s;
hipsparseHandle_t handle_ss;
hipsparseMatDescr_t descr;

// ====== Helpers ======

const char* cublasGetErrorString(hipblasStatus_t status) {
  switch(status) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  return "unknown error";
}

// TODO
const char* hipsparseGetErrorString(hipsparseStatus_t status) {
  return "cusparse error";
}

#define CUDA_CALL(func) { \
  hipError_t e = (func); \
  if(e != hipSuccess) {\
    cout << "CUDA: " << hipGetErrorString(e) << endl; \
    assert(false);\
  }\
}

#define CUBLAS_CALL(func) {\
  hipblasStatus_t e = (func); \
  if(e != HIPBLAS_STATUS_SUCCESS) {\
    cout << "cuBlas: " << cublasGetErrorString(e) << endl; \
    assert(false);\
  }\
}

#define CUSPARSE_CALL(func) {\
  hipsparseStatus_t e = (func); \
  if(e != HIPSPARSE_STATUS_SUCCESS) {\
    cout << "cusparse: " << hipsparseGetErrorString(e) << endl; \
    assert(false);\
  }\
}

// ===== functional =====

bool
staySparse(int n, int nnz) {
  if (nnz < n * n * MAGIC_SPARSE_THRESHOLD1) {
    return true;
  } else {
    return false;
  }
}

void
countNNZ(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        int* nnzrow, int &nnz_total, float* gpu_m, int n) {
  CUSPARSE_CALL(
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, n,
               n, descr,
               gpu_m,
               n, nnzrow, &nnz_total)
  );
  CUDA_CALL(hipDeviceSynchronize());
  cout << "  count nnz [nnz_total=" << nnz_total << "]\n";
}

int
countResultNNZ(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        float *csr_val, int *csr_rowptr, int *csr_colind,
        int* &csr_rowptr_c,
        int nnz_total, int n) {
  int baseC, nnzC;
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzC;
  CUSPARSE_CALL(hipsparseXcsrgemmNnz(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        n, n, n,
        descr, nnz_total, csr_rowptr, csr_colind,
        descr, nnz_total, csr_rowptr, csr_colind,
        descr, csr_rowptr_c, nnzTotalDevHostPtr ));
  if (NULL != nnzTotalDevHostPtr){
      nnzC = *nnzTotalDevHostPtr;
  } else {
      CUDA_CALL(hipMemcpy(&nnzC, csr_rowptr_c+n, sizeof(int), hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy(&baseC, csr_rowptr_c, sizeof(int), hipMemcpyDeviceToHost));
      nnzC -= baseC;
  }
  return nnzC;
}

void
cudaDense2sparse(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        float* gpu_m, int *nnz_row,
        float* &csr_val, int* &csr_rowptr, int* &csr_colind,
        int nnz_total, int n) {
  CUSPARSE_CALL(
    hipsparseSdense2csr(handle, n, n,
               descr,
               gpu_m,
               n, nnz_row,
               csr_val,
               csr_rowptr, csr_colind)
   );
  CUDA_CALL(hipDeviceSynchronize());
  cout << "  [GPU] dense matrix => sparse matrix \n";
}


void
sparse2dense(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        float* csr_val, int* csr_rowptr, int* csr_colind,
        float* gpu_m, int n) {
  CUSPARSE_CALL(
    hipsparseScsr2dense(handle, n, n,
               descr,
               csr_val, csr_rowptr, csr_colind,
               gpu_m,
               n)
   );
  CUDA_CALL(hipDeviceSynchronize());
  cout << "  [GPU] sparse matrix => dense matrix \n";
}

void
sparseSmm(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        float *csr_val, int *csr_rowptr, int *csr_colind,
        float *gpu_src, float *gpu_dst,
        int nnz_total, int n) {
  CUSPARSE_CALL(hipsparseScsrmm(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      n, n, n, nnz_total,
      &alpha, descr,
      csr_val, csr_rowptr, csr_colind,
      gpu_src, n,
      &beta, gpu_dst, n));
  CUDA_CALL(hipDeviceSynchronize());
  cout << "  [GPU] sparse mm\n";
}

void
denseSgemm(hipblasHandle_t handle, float *gpu_src, float *gpu_dst, int n) {
  CUBLAS_CALL(hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        gpu_src, n,
        gpu_src, n,
        &beta,
        gpu_dst, n));
  CUDA_CALL(hipDeviceSynchronize());
  cout<< "  [GPU] dense gemm\n";
}

void
denseStrmm(hipblasHandle_t handle, float *gpu_src, float *gpu_dst, int n) {
  CUBLAS_CALL(hipblasStrmm(
      handle,
      HIPBLAS_SIDE_LEFT,
      HIPBLAS_FILL_MODE_UPPER,
      HIPBLAS_OP_N,
      HIPBLAS_DIAG_UNIT,
      n, n,
      &alpha,
      gpu_src, n,
      gpu_src, n,
      gpu_dst, n));
  CUDA_CALL(hipDeviceSynchronize());
  cout<< "  [GPU] dense trmm\n";
}

void
sparseSparseSmm(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
        float *csr_val, int *csr_rowptr, int *csr_colind,
        float *csr_val_c, int *csr_rowptr_c, int *csr_colind_c,
        int nnz_total, int n) {
  CUSPARSE_CALL(hipsparseScsrgemm(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      n, n, n,
      descr, nnz_total,
      csr_val, csr_rowptr, csr_colind,
      descr, nnz_total,
      csr_val, csr_rowptr, csr_colind,
      descr,
      csr_val_c, csr_rowptr_c, csr_colind_c));
  CUDA_CALL(hipDeviceSynchronize());
  cout << "  [GPU] sparse-sparse mm \n";
}

int
dense2sparse(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
  int *nnz_row, float *dense_m,
  float *csr_val, int *csr_rowptr, int *csr_colind,
  int n) {
  int nnz_total;
  // count number of non-zero element
  countNNZ(handle, descr, nnz_row, nnz_total, dense_m, n);
  if (nnz_total > MAX_NNZ) {
    cout << "[INFO] too many non-zeros(" << nnz_total << "), maximum " << MAX_NNZ << "\n";
    cout << "[INFO] stop using sparse\n";
    //assert(false);
  } else {
    // init the sparse matrix
  cudaDense2sparse(handle, descr, dense_m, nnz_row, csr_val,
        csr_rowptr, csr_colind, nnz_total, n);
    cout << "[INFO] matrix is sparse, using sparse\n";
  }
  return nnz_total;
}

int
sparseSparseMM(hipsparseHandle_t handle, hipsparseMatDescr_t descr,
    float* &csr_val, int* &csr_rowptr, int* &csr_colind,
    int nnz, int n) {
  int *csr_rowptr_ret;
  CUDA_CALL(hipMalloc(&csr_rowptr_ret, sizeof(int)*(n+1)));

  // (1) count result nnz
  int nnz_ret = countResultNNZ(handle, descr,
      csr_val, csr_rowptr, csr_colind,
      csr_rowptr_ret, nnz, n);

  // (2) alloc result memory
  float *csr_val_ret;
  int   *csr_colind_ret;
  CUDA_CALL(hipMalloc(&csr_val_ret, sizeof(float)*nnz_ret));
  CUDA_CALL(hipMalloc(&csr_colind_ret, sizeof(int)*nnz_ret));

  // (3) calc!
  sparseSparseSmm(handle, descr,
    csr_val, csr_rowptr, csr_colind,
    csr_val_ret, csr_rowptr_ret, csr_colind_ret,
    nnz, n);

  // (4) free previous resource and swap the poniter
  CUDA_CALL(hipFree(csr_rowptr));
  CUDA_CALL(hipFree(csr_val));
  CUDA_CALL(hipFree(csr_colind));
  csr_rowptr = csr_rowptr_ret;
  csr_val = csr_val_ret;
  csr_colind = csr_colind_ret;

  return nnz_ret;
}

void regulateCPU(float* a, int size) {
  for (int i=0; i<size; i++) {
    a[i] = 2 * (a[i] != 0);
  }
}

__global__
void regulateGPU(float *a, int length) {
  int index = (threadIdx.x + blockIdx.x * blockDim.x) * REGULATE_BATCH;
  //printf("block %d, thread %d, index[%d] => [%f]\n", blockIdx.x, threadIdx.x, index, a[index]);
  for (int i=0; i<REGULATE_BATCH; i++) {
    if (index+i < length) {
      a[index + i] = 2 * (a[index + i] != 0);
    }
  }
}

void regulate(float *gpu_m, int length, float *cpu_m) {
#ifdef REGULATE_GPU
  int num_blocks = ceil((double)length/THREADS_PER_BLOCK/REGULATE_BATCH);
  regulateGPU<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_m, length);
  auto e = hipGetLastError();
  if ( hipSuccess !=  e ) {
    cout << "CUDA: " << hipGetErrorString(e) << endl;
    assert(false);
  }
  CUDA_CALL(hipDeviceSynchronize());
#else
  CUDA_CALL(hipMemcpy(cpu_m, gpu_m, length*sizeof(float), hipMemcpyDeviceToHost));
  regulateCPU(cpu_m, length);
  CUDA_CALL(hipMemcpy(gpu_m, cpu_m, length*sizeof(float), hipMemcpyHostToDevice));
#endif
}


__device__ int matrix_diff;

__global__
void initEarlyTermination() {
  matrix_diff = 0;
}

__global__
void compareGPU(float *gpu_m_1, float *gpu_m_2, int length) {
  int index = (threadIdx.x + blockIdx.x * blockDim.x) * REGULATE_BATCH;
  for (int i=0; i<REGULATE_BATCH; i++) {
    if (index+i < length) {
      if ( (gpu_m_1[index + i] != 0) != (gpu_m_2[index + i] != 0) ) {
        matrix_diff = 1;
      }
    }
  }
}


bool earlyTermination2(float *gpu_m_1, float *gpu_m_2, int length, int dense_m, int sparse_m) {
#ifdef OPT_EARLY_TERMINATION
  if ( ((dense_m > sparse_m) ? dense_m : sparse_m) < MAGIC_EARLY_TERMINATION_THRESHOLD) {
    return false;
  }
  initEarlyTermination<<<1,1>>>();
  auto e = hipGetLastError();
  if ( hipSuccess !=  e ) {
    cout << "CUDA: " << hipGetErrorString(e) << endl;
    assert(false);
  }
  int num_blocks = ceil((double)length/THREADS_PER_BLOCK/REGULATE_BATCH);
  compareGPU<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_m_1, gpu_m_2, length);
  e = hipGetLastError();
  if ( hipSuccess !=  e ) {
    cout << "CUDA: " << hipGetErrorString(e) << endl;
    assert(false);
  }
  CUDA_CALL(hipDeviceSynchronize());
  typeof(matrix_diff) diff;
  hipMemcpyFromSymbol(&diff, HIP_SYMBOL(matrix_diff), sizeof(diff), 0, hipMemcpyDeviceToHost);
  // if they are the same, we're done
  return diff == 0;
#else
  return false;
#endif
}

bool earlyTermination(float *gpu_m_1, float *gpu_m_2, int length) {
#ifdef OPT_EARLY_TERMINATION
  float* result_1 = (float*) malloc (sizeof(float));
  CUBLAS_CALL(hipblasSasum(handle_c, length, gpu_m_1, 1 /*?*/, result_1));
  float* result_2 = (float*) malloc (sizeof(float));
  CUBLAS_CALL(hipblasSasum(handle_c, length, gpu_m_2, 1 /*?*/, result_2));
  if (*result_1 == *result_2) {
    printf("EarlyTermination: %.3f == %.3f\n", *result_1, *result_2);
  }
  return *result_1 == *result_2;
#else
  return false;
#endif
}




void swapSrcDst(float *&gpu_src, float *&gpu_dst) {
  // swap
  float *tmp = gpu_src;
  gpu_src = gpu_dst;
  gpu_dst = tmp;
}

// ====== exposed functions =====

JNIEXPORT void JNICALL Java_gpu_GPUmm_init(JNIEnv *env, jclass cls) {
  int n = MAX_N;
  int nnz_total = MAX_NNZ;

  // (1) allocate and initialize GPU matrix memory
  CUBLAS_CALL(hipblasCreate(&handle_c));
  CUDA_CALL(hipMalloc(&gpu_m, n*n*sizeof(float)));
  CUDA_CALL(hipMalloc(&gpu_m2, n*n*sizeof(float)));

  // (2) decide whether use sparse matrix
  //     if so, allocate sparse matrix memory
  CUSPARSE_CALL(hipsparseCreate(&handle_s));
  CUSPARSE_CALL(hipsparseCreate(&handle_ss));
  CUSPARSE_CALL(hipsparseSetPointerMode(handle_ss, HIPSPARSE_POINTER_MODE_HOST));
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUDA_CALL(hipMalloc(&gpu_nnz_row, sizeof(int) * n));
  CUDA_CALL(hipMalloc(&gpu_csr_val, sizeof(float) * nnz_total)  );
  CUDA_CALL(hipMalloc(&gpu_csr_rowptr, sizeof(int) * (n+1) ) ) ;
  CUDA_CALL(hipMalloc(&gpu_csr_colind, sizeof(int) * nnz_total) ) ;
}

JNIEXPORT void JNICALL Java_gpu_GPUmm_destroy(JNIEnv *env, jclass cls) {
  CUDA_CALL(hipFree(gpu_m));
  CUDA_CALL(hipFree(gpu_m2));
  CUDA_CALL(hipFree(gpu_nnz_row));
  CUDA_CALL(hipFree(gpu_csr_val));
  CUDA_CALL(hipFree(gpu_csr_rowptr));
  CUDA_CALL(hipFree(gpu_csr_colind));
}


void dumpM(float* a, int n);
/*
 * Connect src_list -> dst_list and update the reachability matrix
 */
JNIEXPORT void JNICALL Java_gpu_GPUmm_connect(JNIEnv *env, jclass cls,
    jfloatArray fb, jintArray src_list, jintArray dst_list, jint jn)
{
  //cout << "    [GPU] gpu connect start...\n";
  int n = (int) jn;
  int len = (int) env->GetArrayLength(src_list);
  int m_size = sizeof(float) * n * len;
  int src_inds[len], dst_inds[len];

  jint *jsrc_inds = env->GetIntArrayElements(src_list, 0);
  jint *jdst_inds = env->GetIntArrayElements(dst_list, 0);
  for (int i=0; i<len; i++) {
    src_inds[i] = jsrc_inds[i];
    dst_inds[i] = jdst_inds[i];
  }

  // FIXME: can reuse some of the other matrix space
  float *cpu_src_matrix, *cpu_dst_matrix, *gpu_src_matrix, *gpu_dst_matrix;
  cpu_src_matrix = (float*) malloc(m_size);
  cpu_dst_matrix = (float*) malloc(m_size);
  CUDA_CALL(hipMalloc(&gpu_src_matrix, m_size));
  CUDA_CALL(hipMalloc(&gpu_dst_matrix, m_size));

  float *cpu_matrix = (float*) env->GetPrimitiveArrayCritical(fb, 0);
  if (cpu_matrix == NULL) {
    cout << "cpu_matrix is NULL!!!\n";
    return;
  }

  // update connect nodes
  // This should happen before collecting the update matrix
  for (int i=0; i<len; i++) {
    int src = src_inds[i];
    int dst = dst_inds[i];
    // NOTE: this is COLUMN-MAJOR storage!!!!
    cpu_matrix[dst*n + src] = 1;
    // UTBABUG: src->[dst~->all] and [all~->src]->dst
    for (int j=0; j<n; j++) {
      // src->[dst~->all]
      if (cpu_matrix[j*n + dst] != 0) {
        cpu_matrix[j*n + src] = 1;
      }
      // [all~->src]->dst
      if (cpu_matrix[src*n + j] != 0) {
        cpu_matrix[dst*n + j] = 1;
      }
    }
  }

  // construct update matrix
  for (int i=0; i<len; i++) {
    int src = src_inds[i];
    int dst = dst_inds[i];
    // NOTE: this is COLUMN-MAJOR storage!!!!
    // vector[x->src]
    for (int j=0; j<n; j++) {
      //cout << "src---["<<i*n+j<<"/"<<m_size<<"], [" << src*n +j << "/" << n*n << "]\n";
      cpu_src_matrix[i*n + j] = cpu_matrix[src*n + j];
    }
    // vector[dst->x]
    for (int j=0; j<n; j++) {
      //cout << "dst---["<<j*len+i<<"/"<<m_size<<"], [" << j*n+dst << "/" << n*n << "]\n";
      cpu_dst_matrix[j*len + i] = cpu_matrix[j*n + dst];
    }
  }

  CUDA_CALL(hipMemcpy(gpu_src_matrix, cpu_src_matrix, m_size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(gpu_dst_matrix, cpu_dst_matrix, m_size, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(gpu_m, cpu_matrix, n*n*sizeof(float), hipMemcpyHostToDevice));

  const float m_beta = 1.0;
  // core: A x B + C -> C
  CUBLAS_CALL(hipblasSgemm(
        handle_c,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, len,
        &alpha,
        gpu_src_matrix, n,
        gpu_dst_matrix, len,
        &m_beta,
        gpu_m, n));
  CUDA_CALL(hipDeviceSynchronize());

  // regulate the matrix
  CUDA_CALL(hipMemcpy(cpu_matrix, gpu_m, n*n*sizeof(float), hipMemcpyDeviceToHost));
  regulateCPU(cpu_matrix, n*n);

  // done
  env->ReleasePrimitiveArrayCritical(fb, cpu_matrix, 0);

  // free GPU memory
  CUDA_CALL(hipFree(gpu_src_matrix));
  CUDA_CALL(hipFree(gpu_dst_matrix));
  free(cpu_src_matrix);
  free(cpu_dst_matrix);

  //cout << "  [GPU] ...connect ends\n";
}



void dumpPartM(float* a, int printn, int n);

int
power(float *cpu_m, int n, bool fresh) {
  if (n > MAX_N) {
    cout << "ERROR, too large a 'n'(" << n << ") size, maximum " << MAX_N << "\n";
    assert(false);
  }
  cout << "[INFO] n=" << n << "\n";


  // (1) copy the matrix to GPU
  CUDA_CALL(hipMemcpy(gpu_m, cpu_m, n*n*sizeof(float), hipMemcpyHostToDevice));

  // (2) check if to use sparse
  int nnz = fresh ? dense2sparse(handle_s, descr, gpu_nnz_row, gpu_m,
               gpu_csr_val, gpu_csr_rowptr, gpu_csr_colind, n) :
            MAX_NNZ;

  // (3) matrix multiplication
  timeval start, end;
  gettimeofday(&start, 0);

  int dense_m = 1;
  // (3.1) sparse mm first
  bool used_sparse = false;
  while(fresh && staySparse(n, nnz)) {
    nnz = sparseSparseMM(handle_ss, descr,
      gpu_csr_val, gpu_csr_rowptr, gpu_csr_colind, nnz, n);
    regulate(gpu_csr_val, nnz, cpu_m);
    dense_m*= 2;
    used_sparse = true;
  }

  // (3.2) convert sparse to dense
  if (used_sparse) {
    sparse2dense(handle_s, descr,
        gpu_csr_val, gpu_csr_rowptr, gpu_csr_colind,
        gpu_m, n);
    // reset the memory used by sparse MM
    CUDA_CALL(hipFree(gpu_csr_val));
    CUDA_CALL(hipFree(gpu_csr_rowptr));
    CUDA_CALL(hipFree(gpu_csr_colind));
    CUDA_CALL(hipMalloc(&gpu_csr_val, sizeof(float) * MAX_NNZ)  );
    CUDA_CALL(hipMalloc(&gpu_csr_rowptr, sizeof(int) * (MAX_N +1) ) ) ;
    CUDA_CALL(hipMalloc(&gpu_csr_colind, sizeof(int) * MAX_NNZ) ) ;
  }

  // (3.3) dense mm then
  float *gpu_src = gpu_m;
  float *gpu_dst = gpu_m2;

  while(dense_m < n) {
#ifdef OPT_TRIANGULAR_MM
    denseStrmm(handle_c, gpu_src, gpu_dst, n);
#else
    denseSgemm(handle_c, gpu_src, gpu_dst, n);
#endif
    dense_m *= 2;
    regulate(gpu_dst, n*n, cpu_m);
    if(earlyTermination(gpu_src, gpu_dst, n*n)) {
      cout << "Early termination, dense_m=" << dense_m << ", n=" << n << "\n";
      break;
    }
    swapSrcDst(gpu_src, gpu_dst);
  }

  gettimeofday(&end, 0);
  double milli = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec) * .001;

  // (4) copy the result out
  CUDA_CALL(hipMemcpy(cpu_m, gpu_m, n*n*sizeof(float), hipMemcpyDeviceToHost));
  cout << "DONE, DM^" << dense_m << ", time = " << milli << "ms\n";

  return 0;
}



JNIEXPORT void JNICALL Java_gpu_GPUmm_power (JNIEnv *env, jclass cls, jfloatArray jarr, jint jn, jboolean jfresh) {
  int n = (int) jn;
  bool fresh = (bool) jfresh;
  float *matrix = (float*) env->GetPrimitiveArrayCritical(jarr, 0);
  //float *matrix = (float*) env->GetFloatArrayElements(jarr, 0);
  if (matrix == NULL) {
    cout << "NULL!!!\n";
    return;
  }

  power(matrix, n, fresh);

  /*
  // debug code
  ofstream outf;
  outf.open("/tmp/mmresult");
  for(int i=0; i<n*n; i++) {
    if (matrix[i] != 0) {
      outf << "1";
    } else {
      outf << "0";
    }
  }
  outf.close();
  */

  env->ReleasePrimitiveArrayCritical(jarr, matrix, 0);
  //env->ReleaseFloatArrayElements(jarr, matrix, 0);
}


int
selfmm(float *cpu_m, int n) {
  if (n > MAX_N) {
    cout << "ERROR, selfmm, too large a 'n'(" << n << ") size, maximum " << MAX_N << "\n";
    assert(false);
  }
  cout << "[INFO] selfmm, n=" << n << "\n";

  float *gpu_src = gpu_m;
  float *gpu_dst = gpu_m2;

  // (1) copy the matrix to GPU
  CUDA_CALL(hipMemcpy(gpu_src, cpu_m, n*n*sizeof(float), hipMemcpyHostToDevice));

  // (3) matrix multiplication
  timeval start, end;
  gettimeofday(&start, 0);

#ifdef OPT_TRIANGULAR_MM
        denseStrmm(handle_c, gpu_src, gpu_dst, n);
#else
        denseSgemm(handle_c, gpu_src, gpu_dst, n);
#endif

  gettimeofday(&end, 0);
  double milli = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec) * .001;

  // (4) copy the result out
  CUDA_CALL(hipMemcpy(cpu_m, gpu_dst, n*n*sizeof(float), hipMemcpyDeviceToHost));
  cout << "DONE, selfmm, time = " << milli << "ms\n";

  return 0;
}


JNIEXPORT void JNICALL Java_gpu_GPUmm_selfmm(JNIEnv *env, jclass cls, jfloatArray jarr, jint jn) {
  int n = (int) jn;
  float *matrix = (float*) env->GetPrimitiveArrayCritical(jarr, 0);
  //float *matrix = (float*) env->GetFloatArrayElements(jarr, 0);
  if (matrix == NULL) {
    cout << "NULL!!!\n";
    return;
  }

  selfmm(matrix, n);

  env->ReleasePrimitiveArrayCritical(jarr, matrix, 0);
  //env->ReleaseFloatArrayElements(jarr, matrix, 0);
}

void dumpM(float* a, int n) {
  cout << "=== n=" << n <<"\n";
  for (int i=0; i<n; i++) {
    for (int j=0; j<n; j++) {
      cout << a[i*n+j] << "  ";
    }
    cout << "\n";
  }
  cout << "===\n";
}

void dumpPartM(float* a, int printn, int n) {
  cout << "=== n=" << n <<"\n";
  for (int i=n/2; i<n/2+printn; i++) {
    for (int j=n/2; j<n/2+printn; j++) {
      cout << a[i*n+j] << "  ";
    }
    cout << "\n";
  }
  cout << "===\n";
}
